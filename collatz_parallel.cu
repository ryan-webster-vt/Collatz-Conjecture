#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

typedef unsigned int uint32;
typedef unsigned long long int uint64;

__device__ int collatz_iter(uint64 start, bool* thread_overflow) {
    uint32 total = 0;
    uint64 a_i = start;
    while (a_i != 1) {
        total++;
        if (a_i % 2 == 0) {
            a_i /= 2;
        } else {
            if (a_i <= 6148914691236517204ULL) {
                a_i = 3 * a_i + 1;
            } else {
                *thread_overflow = true;
                break;
            }
        }
    }
    return total;
}

__device__ uint64 compress_info(int thread_total, uint64 start) {
    uint64 info = thread_total;
    info = info << 48;
    info += start;
    return info;
}


__global__ void collatz(uint64 m, uint64 n, uint64* max_info, int* total_overflow) {
    uint64 thread_num = (uint64)blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_num < (uint64) n - m + 1) {
        uint64 value = m + thread_num;
        bool thread_overflow = false;
        int thread_total = collatz_iter(value, &thread_overflow);
        if (thread_overflow) {
            atomicAdd(total_overflow, 1);
        }
        uint64 info = compress_info(thread_total, value);
        atomicMax(max_info, info);
    }
}

void extract_info (uint64 info, int* thread_total, uint64* start) {
    *start = info & 0xFFFFFFFFFFFF;
    info = info >> 48;
    *thread_total = info;
}

int main (int argc, char** argv) {

    // get M and N from the command line
    if (argc < 4) {
        printf ("Command usage : %s %s %s %s\n",argv[0],"M","N","B");
        return 1;
    }
    uint64 M = atol(argv[1]);
    uint64 N = atol(argv[2]);
    uint64 B = atol(argv[3]);

    // start the timer
    clock_t start = clock();

    uint64* d_max_info;
    hipMalloc(&d_max_info, sizeof(uint64));
    if (d_max_info == NULL) {
        printf("hipMalloc failed\n");
        return 1;
    }

    int* d_total_overflow;
    hipMalloc(&d_total_overflow, sizeof(int));
    if (d_total_overflow == NULL) {
        printf("hipMalloc failed\n");
        return 1;
    }

    hipMemset(d_max_info, 0, sizeof(uint64));
    hipMemset(d_total_overflow, 0, sizeof(int));

    int G = (N - M + B - 1) / B;
    collatz<<<G, B>>>(M, N, d_max_info, d_total_overflow);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize(); // Ensure kernel execution has finished

    int num_overflow;
    uint64 max_info;
    hipMemcpy(&max_info, d_max_info, sizeof(uint64), hipMemcpyDeviceToHost);
    hipMemcpy(&num_overflow, d_total_overflow, sizeof(int), hipMemcpyDeviceToHost);

    // stop the timer
    clock_t stop = clock();
    double elapsed = (double)(stop-start)/CLOCKS_PER_SEC;

    int max_total;
    uint64 max_start;
    extract_info(max_info, &max_total, &max_start);

    // output the results
    printf("M = %llu, N = %llu\n", M, N);
    printf("threads per block B = %llu\n", B);
    printf("number of thread blocks G = %u\n", G);
    printf("number of threads G*B = %llu\n", G*B);
    printf ("A starting value between %llu and %llu (inclusive)\n",M,N);
    printf ("  having the largest total stopping time is %llu\n",max_start);
    printf ("  which has %u steps\n",max_total);
    printf ("Number of start values that result in overflow = %d\n",num_overflow);
    printf ("elapsed time = %.2f seconds\n",elapsed);

    hipFree(d_max_info);
    hipFree(d_total_overflow);

    return 0;
}
